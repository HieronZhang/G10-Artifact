#include <assert.h>
#include <vector>
#include "Dropout_Forward.h"

Dropout_Forward::Dropout_Forward(hipdnnHandle_t handle, vector<double> &args, bool is_UVM) : 
        handle(handle), is_UVM(is_UVM) {
    // 0. batch_size    1. in_channels   2. input_height    3. input_width
    input_n    = args[0];   input_c    = args[1];   input_h    = args[2];   input_w  = args[3];
    input_ratio = args[4]; output_ratio = args[5];

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreateDropoutDescriptor(&dropout_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));

    // SetInputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // SetDropoutDescriptor
    CUDNN_CALL(hipdnnDropoutGetStatesSize(
            handle, &state_size));
    CUDA_CALL(hipMalloc(&state_data, state_size));
    CUDNN_CALL(hipdnnSetDropoutDescriptor(
            dropout_descriptor,
            handle,
            dropout,
            state_data,
            state_size,
            seed));
    // SetOutputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // AllocMemory
    CUDNN_CALL(cudnnDropoutGetReserveSpaceSize(
            input_descriptor, &workspace_size));
    // Alloc
    if (!is_UVM) {
        CUDA_CALL(hipMalloc(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&workspace_data, workspace_size));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
    }
    hipDeviceSynchronize();
}

Dropout_Forward::~Dropout_Forward() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyDropoutDescriptor(dropout_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    if (!is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
        CUDA_CALL(hipFree(workspace_data));
        CUDA_CALL(hipFree(state_data));
    }
}

float Dropout_Forward::Run() {
    if (is_UVM) {
        CUDA_CALL(hipMallocManaged(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&workspace_data, workspace_size));
        CPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));

        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(output_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * output_ratio);
        GPUFillRand(workspace_data, workspace_size * output_ratio);
        hipDeviceSynchronize();
    }

    float milliseconds = 0;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start));
    CUDNN_CALL(cudnnDropoutForward(
            handle,
            dropout_descriptor,
            input_descriptor, input_data,
            output_descriptor, output_data,
            workspace_data, workspace_size));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    
    if (is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
        CUDA_CALL(hipFree(workspace_data));
    }
    return milliseconds;
}

size_t Dropout_Forward::getWorkspaceSize() {
    return workspace_size;
}
