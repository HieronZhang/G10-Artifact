#include <assert.h>
#include <vector>
#include "BatchNorm2d_Backward.h"

BatchNorm2d_Backward::BatchNorm2d_Backward(hipdnnHandle_t handle, vector<double> &args, bool is_UVM) : 
        handle(handle), is_UVM(is_UVM) {
    // 0. batch_size    1. in_channels   2. input_height    3. input_width
    input_n    = args[0];   input_c    = args[1];   input_h    = args[2];   input_w  = args[3];
    input_ratio = args[4]; output_ratio = args[5];

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&batch_norm_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&d_input_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&d_output_descriptor));

    // SetInputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            d_output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // SetBatchNormDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            batch_norm_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            1, input_c, 1, 1));
    // SetOutputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            d_input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // Alloc
    if (!is_UVM) {
        CUDA_CALL(hipMalloc(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&bn_scale, (long) input_c * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_bn_scale, (long) input_c * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_bn_bias, (long) input_c * sizeof(float)));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(d_output_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(bn_scale, (long) input_c * sizeof(float));
        GPUFillRand(d_bn_scale, (long) input_c * sizeof(float));
        GPUFillRand(d_bn_bias, (long) input_c * sizeof(float));
    }
    hipDeviceSynchronize();
}

BatchNorm2d_Backward::~BatchNorm2d_Backward() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(batch_norm_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(d_input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(d_output_descriptor));
    if (!is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(d_input_data));
        CUDA_CALL(hipFree(d_output_data));
        CUDA_CALL(hipFree(d_bn_scale));
        CUDA_CALL(hipFree(d_bn_bias));
    }
}

float BatchNorm2d_Backward::Run() {
    if (is_UVM) {
        CUDA_CALL(hipMallocManaged(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&d_output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&bn_scale, (long) input_c * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&d_bn_scale, (long) input_c * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&d_bn_bias, (long) input_c * sizeof(float)));
        CPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        CPUFillRand(d_output_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        CPUFillRand(bn_scale, (long) input_c * sizeof(float));
        CPUFillRand(d_bn_scale, (long) input_c * sizeof(float));
        CPUFillRand(d_bn_bias, (long) input_c * sizeof(float));

        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(d_output_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * output_ratio);
        GPUFillRand(bn_scale, (long) input_c * sizeof(float) * input_ratio);
        GPUFillRand(d_bn_scale, (long) input_c * sizeof(float) * output_ratio);
        GPUFillRand(d_bn_bias, (long) input_c * sizeof(float) * output_ratio);
        hipDeviceSynchronize();
    }

    float one = 1;
    float zero = 0;
    
    float milliseconds = 0;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start));
    hipdnnBatchNormalizationBackward(
        handle,
        HIPDNN_BATCHNORM_SPATIAL,/* hipdnnBatchNormMode_t mode */
        &one,
        &zero,
        &one,
        &zero,
        input_descriptor,       /* const hipdnnTensorDescriptor_t xDesc */
        input_data,             /* const void *x */
        d_output_descriptor,      /* const hipdnnTensorDescriptor_t yDesc */
        d_output_data,            /* void *y */
        d_input_descriptor,
        d_input_data,
        batch_norm_descriptor,  /* const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc */
        bn_scale,
        d_bn_scale,
        d_bn_bias,
        epsilon,
        nullptr,
        nullptr);
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

    if (is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(d_input_data));
        CUDA_CALL(hipFree(d_output_data));
        CUDA_CALL(hipFree(d_bn_scale));
        CUDA_CALL(hipFree(d_bn_bias));
    }
    return milliseconds;
}
