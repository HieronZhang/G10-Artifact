#include <assert.h>
#include <vector>
#include "MaxPool2d_Backward.h"

MaxPool2d_Backward::MaxPool2d_Backward(hipdnnHandle_t handle, vector<double> &args, bool is_UVM) : 
        handle(handle), is_UVM(is_UVM) {
    // 0. batch_size    1. in_channels   2. input_height  3. input_width
    // 4. kernel_size_h 5. kernel_size_w
    // 6. padding_h     7. padding_w     8. stride_h      9. stride_w
    input_n    = args[0];   input_c    = args[1];   input_h    = args[2];   input_w    = args[3];
    k_size_h   = args[4];   k_size_w   = args[5];   
    padding_h  = args[6];   padding_w  = args[7];   stride_h   = args[8];   stride_w   = args[9]; 
    input_ratio = args[10]; output_ratio = args[11];

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&d_input_descriptor));
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&maxpool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(
            maxpool_descriptor,
            HIPDNN_POOLING_MAX,
            HIPDNN_NOT_PROPAGATE_NAN,
            k_size_h, k_size_w,
            padding_h, padding_w,
            stride_h, stride_w));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&d_output_descriptor));

    // SetInputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            d_input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // SetOutputDescriptor
    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
            maxpool_descriptor, 
            input_descriptor,
            &output_n, &output_c, &output_h, &output_w));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            output_n, output_c, output_h, output_w));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            d_output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            output_n, output_c, output_h, output_w));
    // Alloc
    if (!is_UVM) {
        CUDA_CALL(hipMalloc(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&output_data, (long) output_n * output_c * output_h * output_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float)));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(output_data, (long) output_n * output_c * output_h * output_w * sizeof(float));
        GPUFillRand(d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float));
    }
    hipDeviceSynchronize();
}

MaxPool2d_Backward::~MaxPool2d_Backward() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(d_input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(d_output_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(maxpool_descriptor));
    if (!is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
        CUDA_CALL(hipFree(d_output_data));
    }
}

float MaxPool2d_Backward::Run() {
    if (is_UVM) {
        CUDA_CALL(hipMallocManaged(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&output_data, (long) output_n * output_c * output_h * output_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float)));
        CPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        CPUFillRand(output_data, (long) output_n * output_c * output_h * output_w * sizeof(float));
        CPUFillRand(d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float));

        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * output_ratio);
        GPUFillRand(d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float) * input_ratio);
        hipDeviceSynchronize();
    }

    float milliseconds = 0;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start));
    CUDNN_CALL(hipdnnPoolingBackward(
            handle,
            maxpool_descriptor,
            &alpha,
            output_descriptor, output_data,
            d_output_descriptor, d_output_data,
            input_descriptor, input_data,
            &beta,
            d_input_descriptor, d_input_data));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    
    if (is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(d_input_data));
        CUDA_CALL(hipFree(output_data));
        CUDA_CALL(hipFree(d_output_data));
    }
    return milliseconds;
}
