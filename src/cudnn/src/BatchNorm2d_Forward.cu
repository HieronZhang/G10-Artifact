#include <assert.h>
#include <vector>
#include "BatchNorm2d_Forward.h"

BatchNorm2d_Forward::BatchNorm2d_Forward(hipdnnHandle_t handle, vector<double> &args, bool is_UVM) : 
        handle(handle), is_UVM(is_UVM) {
    // 0. batch_size    1. in_channels   2. input_height    3. input_width
    input_n    = args[0];   input_c    = args[1];   input_h    = args[2];   input_w  = args[3];
    input_ratio = args[4]; output_ratio = args[5];

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&batch_norm_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));

    // SetInputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // SetBatchNormDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            batch_norm_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            1, input_c, 1, 1));
    // SetOutputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // Alloc
    if (!is_UVM) {
        CUDA_CALL(hipMalloc(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&result_running_mean, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&result_running_variance, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&bn_scale, (long) input_c * sizeof(float)));
        CUDA_CALL(hipMalloc(&bn_bias, (long) input_c * sizeof(float)));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(result_running_mean, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(result_running_variance, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(bn_scale, (long) input_c * sizeof(float));
        GPUFillRand(bn_bias, (long) input_c * sizeof(float));
    }
    hipDeviceSynchronize();
}

BatchNorm2d_Forward::~BatchNorm2d_Forward() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(batch_norm_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    if (!is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
        CUDA_CALL(hipFree(result_running_mean));
        CUDA_CALL(hipFree(result_running_variance));
        CUDA_CALL(hipFree(bn_scale));
        CUDA_CALL(hipFree(bn_bias));
    }
}

float BatchNorm2d_Forward::Run() {
    if (is_UVM) {
        CUDA_CALL(hipMallocManaged(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&result_running_mean, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&result_running_variance, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&bn_scale, (long) input_c * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&bn_bias, (long) input_c * sizeof(float)));
        CPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        CPUFillRand(result_running_mean, (long) input_n * input_c * input_h * input_w * sizeof(float));
        CPUFillRand(result_running_variance, (long) input_n * input_c * input_h * input_w * sizeof(float));
        CPUFillRand(bn_scale, (long) input_c * sizeof(float));
        CPUFillRand(bn_bias, (long) input_c * sizeof(float));

        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(result_running_mean, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(result_running_variance, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(output_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * output_ratio);
        GPUFillRand(bn_scale, (long) input_c * sizeof(float) * input_ratio);
        GPUFillRand(bn_bias, (long) input_c * sizeof(float) * input_ratio);
        hipDeviceSynchronize();
    }

    float one = 1;
    float zero = 0;
    
    float milliseconds = 0;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start));
    CUDNN_CALL(hipdnnBatchNormalizationForwardTraining(
        handle,
        HIPDNN_BATCHNORM_SPATIAL,/* hipdnnBatchNormMode_t mode */
        &one,
        &zero,
        input_descriptor,       /* const hipdnnTensorDescriptor_t xDesc */
        input_data,             /* const void *x */
        output_descriptor,      /* const hipdnnTensorDescriptor_t yDesc */
        output_data,            /* void *y */
        batch_norm_descriptor,  /* const hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc */
        bn_scale,
        bn_bias,
        exponentialAverageFactor,
        result_running_mean,
        result_running_variance,
        epsilon,
        nullptr,
        nullptr));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    
    if (is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
        CUDA_CALL(hipFree(result_running_mean));
        CUDA_CALL(hipFree(result_running_variance));
        CUDA_CALL(hipFree(bn_scale));
        CUDA_CALL(hipFree(bn_bias));
    }
    return milliseconds;
}
