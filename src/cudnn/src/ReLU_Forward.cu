#include <assert.h>
#include <vector>
#include "ReLU_Forward.h"

ReLU_Forward::ReLU_Forward(hipdnnHandle_t handle, vector<double> &args, bool is_UVM) : 
        handle(handle), is_UVM(is_UVM) {
    // 0. batch_size    1. in_channels   2. input_height    3. input_width
    input_n    = args[0];   input_c    = args[1];   input_h    = args[2];   input_w  = args[3];
    input_ratio = args[4]; output_ratio = args[5];

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreateActivationDescriptor(&activation_descriptor));
    CUDNN_CALL(hipdnnSetActivationDescriptor(activation_descriptor,
                                            HIPDNN_ACTIVATION_RELU,
                                            HIPDNN_PROPAGATE_NAN,
                                            0 /* RELU_coef */));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));

    // SetInputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // SetOutputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // Alloc
    if (!is_UVM) {
        CUDA_CALL(hipMalloc(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
    }

    hipDeviceSynchronize();
}

ReLU_Forward::~ReLU_Forward() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    if (!is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
    }
}

float ReLU_Forward::Run() {
    if (is_UVM) {
        CUDA_CALL(hipMallocManaged(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(output_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * output_ratio);
        hipDeviceSynchronize();
    }

    float one = 1;
    float zero = 0;
    
    float milliseconds = 0;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start));
    CUDNN_CALL(hipdnnActivationForward(
            handle,
            activation_descriptor,
            &one,
            input_descriptor,
            input_data,
            &zero,
            output_descriptor,
            output_data));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    
    if (is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
    }
    return milliseconds;
}



Softmax_Forward::Softmax_Forward(hipdnnHandle_t handle, vector<double> &args, bool is_UVM) : 
        handle(handle), is_UVM(is_UVM) {
    // 0. batch_size    1. in_channels   2. input_height    3. input_width
    input_n    = args[0];   input_c    = args[1];   input_h    = args[2];   input_w  = args[3];
    input_ratio = args[4]; output_ratio = args[5];

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));

    // SetInputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // SetOutputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // Alloc
    if (!is_UVM) {
        CUDA_CALL(hipMalloc(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
    }

    hipDeviceSynchronize();
}

Softmax_Forward::~Softmax_Forward() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    if (!is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
    }
}

float Softmax_Forward::Run() {
    if (is_UVM) {
        CUDA_CALL(hipMallocManaged(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&output_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(output_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * output_ratio);
        hipDeviceSynchronize();
    }

    float one = 1;
    float zero = 0;
    
    float milliseconds = 0;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start));
    CUDNN_CALL(hipdnnSoftmaxForward(
            handle,
            algorithm,
            mode,
            &one,
            input_descriptor,
            input_data,
            &zero,
            output_descriptor,
            output_data));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    
    if (is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
    }
    return milliseconds;
}
