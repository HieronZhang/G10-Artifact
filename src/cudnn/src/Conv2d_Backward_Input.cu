#include <assert.h>
#include <vector>
#include "Conv2d_Backward_Input.h"

Conv2d_Backward_Input::Conv2d_Backward_Input(hipdnnHandle_t handle, vector<double> &args, bool is_UVM) : 
        handle(handle), is_UVM(is_UVM) {
    // 0. batch_size    1. in_channels   2. input_height  3. input_width
    // 4. out_channels  5. kernel_size_r 6. kernel_size_s
    // 7. padding0      8. padding1      9. stride0      10. stride1       
    input_n    = args[0];   input_c    = args[1];   input_h    = args[2];   input_w  = args[3];
    filter_n   = args[4];   filter_c   = args[1];   filter_h   = args[5];   filter_w = args[6];
    padding_h  = args[7];   padding_w  = args[8];
    stride_h   = args[9];   stride_w   = args[10];
    input_ratio = args[11]; output_ratio = args[12];
    dilation_h = 1;         dilation_w = 1;

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&d_output_descriptor));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&d_input_descriptor));
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));

    // SetInputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            d_input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // SetFilterDescriptor
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(
            filter_descriptor,
            HIPDNN_DATA_FLOAT,
            HIPDNN_TENSOR_NCHW,
            filter_n, filter_c, filter_h, filter_w));
    // SetConvolutionDescriptor
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
            convolution_descriptor,
            padding_h, padding_w, stride_h, stride_w, dilation_h, dilation_w,
            HIPDNN_CONVOLUTION,
            HIPDNN_DATA_FLOAT));
    // SetOutputDescriptor
    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
            convolution_descriptor,
            input_descriptor, filter_descriptor,
            &output_n, &output_c, &output_h, &output_w));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            d_output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            output_n, output_c, output_h, output_w));
    // SetAlgorithm
    hipdnnConvolutionBwdDataAlgoPerf_t convolution_algo_perf;
    int algo_count;
    cudnnGetConvolutionBackwardDataAlgorithm_v7(
            handle,
            filter_descriptor,
            d_output_descriptor,
            convolution_descriptor,
            d_input_descriptor,
            1,              /*requested algo count*/
            &algo_count,    /*returned algo count*/
            &convolution_algo_perf);

    algorithm = convolution_algo_perf.algo;
    CUDNN_CALL(hipdnnGetConvolutionBackwardDataWorkspaceSize(
            handle,
            filter_descriptor, d_output_descriptor, convolution_descriptor, d_input_descriptor,
            algorithm,
            &workspace_size));
    // Alloc
    if (!is_UVM) {
        CUDA_CALL(hipMalloc(&filter_data, (long) filter_n * filter_c * filter_h * filter_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&workspace_data, workspace_size));
        GPUFillRand(filter_data, (long) filter_n * filter_c * filter_h * filter_w * sizeof(float));
        GPUFillRand(d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float));
        GPUFillRand(workspace_data, workspace_size);
    }

    hipDeviceSynchronize();
}

Conv2d_Backward_Input::~Conv2d_Backward_Input() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(convolution_descriptor));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(d_output_descriptor));
    if (!is_UVM) {
        CUDA_CALL(hipFree(filter_data));
        CUDA_CALL(hipFree(d_output_data));
        CUDA_CALL(hipFree(d_input_data));
        CUDA_CALL(hipFree(workspace_data));
    }
}

float Conv2d_Backward_Input::Run() {
    if (is_UVM) {
        CUDA_CALL(hipMallocManaged(&filter_data, (long) filter_n * filter_c * filter_h * filter_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&workspace_data, workspace_size));
        CPUFillRand(filter_data, (long) filter_n * filter_c * filter_h * filter_w * sizeof(float));
        CPUFillRand(d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float));
        CPUFillRand(workspace_data, workspace_size);
        //std::cout<<"Workspace size:"<<workspace_size<<std::endl;
        GPUFillRand(d_output_data, (long) output_n * output_c * output_h * output_w * sizeof(float) * input_ratio);
        GPUFillRand(filter_data, (long) filter_n * filter_c * filter_h * filter_w * sizeof(float) * input_ratio);
        GPUFillRand(d_input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * output_ratio);
        GPUFillRand(workspace_data, workspace_size * output_ratio);
        hipDeviceSynchronize();
    }

    float milliseconds = 0;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start));
    CUDNN_CALL(hipdnnConvolutionBackwardData(
            handle,
            &alpha,
            filter_descriptor, filter_data,
            d_output_descriptor, d_output_data,
            convolution_descriptor, algorithm, workspace_data, workspace_size,
            &beta,
            d_input_descriptor, d_input_data));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

    if (is_UVM) {
        CUDA_CALL(hipFree(filter_data));
        CUDA_CALL(hipFree(d_output_data));
        CUDA_CALL(hipFree(d_input_data));
        CUDA_CALL(hipFree(workspace_data));
    }
    return milliseconds;
}

size_t Conv2d_Backward_Input::getWorkspaceSize() {
    return workspace_size;
}
