#include "hip/hip_runtime.h"
#include <assert.h>
#include <vector>
#include "Linear_Backward_Input.h"

#define TILE_SZ_A 128
#define TILE_SZ_B 16
#define TILE_SZ_RATIO (TILE_SZ_A/TILE_SZ_B)

__global__ void backward_input(int m, int n, int k, 
        const float *input, const float *weight, float* output) {

    // Macros for accessing flattened matrices
    #define input(row, col)  input[(row) + (col) * m]
    #define weight(row, col) weight[(row) * n + (col)]
    #define output(row, col) output[(row) + (col) * m]

    __shared__ float B_shared[TILE_SZ_RATIO][TILE_SZ_B];
    int row = blockIdx.x * TILE_SZ_A + threadIdx.x;

    int n_iter_num = ceil(n * 1.0 / TILE_SZ_B);
    int k_iter_num = ceil(k * 1.0 / TILE_SZ_RATIO);
    for (int n_iter = 0; n_iter < n_iter_num; n_iter++) {
        for (int k_iter = 0; k_iter < k_iter_num; k_iter++) {
        // load weight tile into shared memory, weight is transposed
        int shared_start_row = k_iter * TILE_SZ_RATIO;
        int shared_start_col = n_iter * TILE_SZ_B;
        int shared_row_offset = threadIdx.x / TILE_SZ_B;
        int shared_col_offset = threadIdx.x % TILE_SZ_B;

        if (shared_start_row + shared_row_offset < k && shared_start_col + shared_col_offset < n) {
            B_shared[shared_row_offset][shared_col_offset] = 
                    weight(shared_start_row + shared_row_offset, shared_start_col + shared_col_offset);
        } else {
            B_shared[shared_row_offset][shared_col_offset] = 0;
        }

        __syncthreads();

        for (int j = 0; j < TILE_SZ_B; j++) {
            float output_cumulative = 0;
            for (int i = 0; i < TILE_SZ_RATIO; i++) {
                if (row < m && shared_start_col + j < n) {
                    output_cumulative += input(row, k_iter * TILE_SZ_RATIO + i) * B_shared[i][j];
                }
            }
            output(row, shared_start_col + j) += output_cumulative;
        }
        
        __syncthreads();
        }
    }
}

Linear_Backward_Input::Linear_Backward_Input(hipdnnHandle_t handle, vector<double> &args, bool is_UVM) : 
        handle(handle), is_UVM(is_UVM) {
    // 0. batch_size    1. in_channels   2. input_height    3. input_width
    // 4. h_in          5. h_out
    input_n    = args[0];   input_c    = args[1];   input_h    = args[2];   input_w  = args[3];
    h_in       = args[4];   h_out      = args[5];
    reshape = (long) input_n * input_c * input_h * input_w / h_in;
    input_ratio = args[6]; output_ratio = args[7];

    if (!is_UVM) {
        CUDA_CALL(hipMalloc(&d_input_data, (long) reshape * h_in * sizeof(float)));
        CUDA_CALL(hipMalloc(&weight_data, (long) h_in * h_out * sizeof(float)));
        CUDA_CALL(hipMalloc(&d_output_data, (long) reshape * h_out * sizeof(float)));
        GPUFillRand(weight_data, (long) h_in * h_out * sizeof(float));
        GPUFillRand(d_output_data, (long) reshape * h_out * sizeof(float));
    }
    hipDeviceSynchronize();
}

Linear_Backward_Input::~Linear_Backward_Input() {
    if (!is_UVM) {
        CUDA_CALL(hipFree(d_input_data));
        CUDA_CALL(hipFree(weight_data));
        CUDA_CALL(hipFree(d_output_data));
    }
}

float Linear_Backward_Input::Run() {
    if (is_UVM) {
        CUDA_CALL(hipMallocManaged(&d_input_data, (long) reshape * h_in * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&weight_data, (long) h_in * h_out * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&d_output_data, (long) reshape * h_out * sizeof(float)));
        CPUFillRand(weight_data, (long) h_in * h_out * sizeof(float));
        CPUFillRand(d_output_data, (long) reshape * h_out * sizeof(float));

        GPUFillRand(weight_data, (long) h_in * h_out * sizeof(float) * input_ratio);
        GPUFillRand(d_output_data, (long) reshape * h_out * sizeof(float) * input_ratio);
        GPUFillRand(d_input_data, (long) reshape * h_in * sizeof(float) * output_ratio);
        hipDeviceSynchronize();
    }

    dim3 BlockSize(TILE_SZ_A, 1, 1);
    dim3 GridSize(ceil((double) reshape / TILE_SZ_A), 1, 1);

    float milliseconds = 0;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start));
    backward_input<<<GridSize, BlockSize>>>(reshape, h_in, h_out, d_output_data, weight_data, d_input_data);
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    
    if (is_UVM) {
        CUDA_CALL(hipFree(d_input_data));
        CUDA_CALL(hipFree(weight_data));
        CUDA_CALL(hipFree(d_output_data));
    }
    return milliseconds;
}
