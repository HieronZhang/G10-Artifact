#include <assert.h>
#include <vector>
#include "AdaptiveAvgPool2d_Forward.h"

AdaptiveAvgPool2d_Forward::AdaptiveAvgPool2d_Forward(hipdnnHandle_t handle, vector<double> &args, bool is_UVM) : 
        handle(handle), is_UVM(is_UVM) {
    // 0. batch_size    1. in_channels   2. input_height  3. input_width
    // 4. kernel_size_h 5. kernel_size_w
    // 6. padding_h     7. padding_w     8. stride_h      9. stride_w
    input_n    = args[0];   input_c    = args[1];   input_h    = args[2];   input_w    = args[3];
    k_size_h   = args[4];   k_size_w   = args[5];   
    padding_h  = args[6];   padding_w  = args[7];   stride_h   = args[8];   stride_w   = args[9]; 
    input_ratio = args[10]; output_ratio = args[11];

    CUDNN_CALL(hipdnnCreateTensorDescriptor(&input_descriptor));
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&avgpool_descriptor));
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(
            avgpool_descriptor,
            HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
            HIPDNN_NOT_PROPAGATE_NAN,
            k_size_h, k_size_w,
            padding_h, padding_w,
            stride_h, stride_w));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&output_descriptor));

    // SetInputDescriptor
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            input_n, input_c, input_h, input_w));
    // SetOutputDescriptor
    CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(
            avgpool_descriptor, 
            input_descriptor,
            &output_n, &output_c, &output_h, &output_w));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
            output_descriptor,
            HIPDNN_TENSOR_NCHW,
            HIPDNN_DATA_FLOAT,
            output_n, output_c, output_h, output_w));
    // Alloc
    if (!is_UVM) {
        CUDA_CALL(hipMalloc(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMalloc(&output_data, (long) output_n * output_c * output_h * output_w * sizeof(float)));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
    }
    hipDeviceSynchronize();
}

AdaptiveAvgPool2d_Forward::~AdaptiveAvgPool2d_Forward() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(input_descriptor));
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(avgpool_descriptor));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(output_descriptor));
    if (!is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
    }
}

float AdaptiveAvgPool2d_Forward::Run() {
    if (is_UVM) {
        CUDA_CALL(hipMallocManaged(&input_data, (long) input_n * input_c * input_h * input_w * sizeof(float)));
        CUDA_CALL(hipMallocManaged(&output_data, (long) output_n * output_c * output_h * output_w * sizeof(float)));
        CPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float));
        GPUFillRand(input_data, (long) input_n * input_c * input_h * input_w * sizeof(float) * input_ratio);
        GPUFillRand(output_data, (long) output_n * output_c * output_h * output_w * sizeof(float) * output_ratio);
        hipDeviceSynchronize();
    }

    float milliseconds = 0;
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    CUDA_CALL(hipEventRecord(start));
    CUDNN_CALL(hipdnnPoolingForward(
            handle,
            avgpool_descriptor,
            &alpha,
            input_descriptor, input_data,
            &beta,
            output_descriptor, output_data));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));

    CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));
    
    if (is_UVM) {
        CUDA_CALL(hipFree(input_data));
        CUDA_CALL(hipFree(output_data));
    }
    return milliseconds;
}
